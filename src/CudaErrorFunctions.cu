#include "hip/hip_runtime.h"
/*
libcudann
Copyright (C) 2011 Luca Donati (lucadonati85@gmail.com)
*/

/*
 * CudaErrorFunctions.cu
 *
 *  Created on: Jan 10, 2011
 *      Author: donati
 */

#include "CudaErrorFunctions.cuh"

#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>


#define BLOCKSIZE 512
#define WARP_SIZE 32



__global__ void error(float * c, const float * a, const float * b, const int number, const int actFunc, const int errorFunc){
	//global thread index
	const int g_tid = BLOCKSIZE * blockIdx.x + threadIdx.x;
	if(g_tid<number){
		float error=(a[g_tid]-b[g_tid])/spanS(actFunc);
		c[g_tid]=calcErr(error,errorFunc);
	}
}
//computes the error function for (number) elements of (a)-(b) and store the results in (c)
void computeError(float * c, const float * a, const float * b, const int number, const int actFunc, const int errorFunc){

int numBlocks = number/BLOCKSIZE+1;
error<<<numBlocks, BLOCKSIZE>>>(c,a,b,number,actFunc,errorFunc);

}


//computes the total mse for (number) elements of (desired)-(neurons)
float mseError(const float * desired, float * neurons, const int number, const int actFunc){

int numBlocks = number/BLOCKSIZE+1;
error<<<numBlocks, BLOCKSIZE>>>(neurons,desired,neurons,number,actFunc,ERROR_LINEAR);
//does the product of each member then sums them all and divides for number
return hipblasSdot(number, neurons, 1, neurons, 1)/(float)number;
}


__global__ void maxes(const int nOfInst, const int nOfOut, const float * neurons, int * indexes){
	extern __shared__ float sdata[];
	float *sidx = sdata + BLOCKSIZE;

	const int tid		= threadIdx.x;					// thread index
	const int thread_lane	= tid & (WARP_SIZE-1);		// thread index within the warp
	const int g_tid		= BLOCKSIZE * blockIdx.x + tid;	// global thread index
	const int g_warp_id	= g_tid / WARP_SIZE;			// global warp index

	const int offset=g_warp_id*nOfOut;

	//loading in shared data of values
	sdata[tid]=((offset+thread_lane<nOfInst*nOfOut)&&(thread_lane<nOfOut))?neurons[offset+thread_lane]:0.0f;
	//loading in shared data of indexes
	sidx[tid]=thread_lane;

	if(g_warp_id<nOfInst){

		//some sequential reduction (suggested to maximize the throughput)
		for(unsigned int i = thread_lane+WARP_SIZE ; i < nOfOut ; i += WARP_SIZE){
			float aux=neurons[offset+i];
			if(sdata[tid] < aux){sdata[tid]=aux;sidx[tid]=i;}
		}

		//parallel reduction of both the value and the index
		if (thread_lane < 16){
			if(sdata[tid] < sdata[tid+16]){sdata[tid]=sdata[tid+16];sidx[tid]=sidx[tid+16];}
			if(sdata[tid] < sdata[tid+8]){sdata[tid]=sdata[tid+8];sidx[tid]=sidx[tid+8];}
			if(sdata[tid] < sdata[tid+4]){sdata[tid]=sdata[tid+4];sidx[tid]=sidx[tid+4];}
			if(sdata[tid] < sdata[tid+2]){sdata[tid]=sdata[tid+2];sidx[tid]=sidx[tid+2];}
			if(sdata[tid] < sdata[tid+1]){sdata[tid]=sdata[tid+1];sidx[tid]=sidx[tid+1];}
		}

		//return the best neuron index
		if (thread_lane == 0){
			indexes[g_warp_id]=sidx[tid];
		}

	}
}
//find the (indexes) of the max values of each row of a set of (neurons), divided in rows(nOfOut) and columns(nOfInst)
void computeMaxes(const int nOfInst, const int nOfOut, const float * neurons, int * indexes){

int numBlocks = nOfInst / (BLOCKSIZE/WARP_SIZE)+1;
int smemSize = 2 * BLOCKSIZE  * sizeof(float);

maxes<<<numBlocks, BLOCKSIZE,smemSize>>>(nOfInst, nOfOut, neurons, indexes);

}



__global__ void addMom(float * weights, float * oldWeights,const int number, const float momentum){
	//global thread index
	const int g_tid = BLOCKSIZE * blockIdx.x + threadIdx.x;
	const float weight=weights[g_tid];
	if(g_tid<number){
		weights[g_tid]+=momentum*(weight-oldWeights[g_tid]);
		oldWeights[g_tid]=weight;
	}
}
//adds to (number) elements of (weights) the difference between (weights) and (oldWeights) multiplied with (momentum). also update (oldWeights)
void addMomentum(float * weights, float * oldWeights,const int number, const float momentum){

int numBlocks = number/BLOCKSIZE+1;

addMom<<<numBlocks, BLOCKSIZE>>>(weights,oldWeights,number,momentum);

}


__global__ void trMatrix(const int x, const int y, const float * in, float * out){
	//global thread index
	const int g_tid = BLOCKSIZE * blockIdx.x + threadIdx.x;

	if(g_tid<x*y){
		out[g_tid%x*y+g_tid/x]=in[g_tid];
	}
}
//translate a matrix x-y (rows large (x) and columns high (y)) to one y-x
void translateMatrix(const int x, const int y, const float * in, float * out){

int numBlocks = (x*y)/BLOCKSIZE+1;

trMatrix<<<numBlocks, BLOCKSIZE>>>(x,y,in,out);

}

